#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

[[gnu::noinline]]
void _abortError(const char* msg, const char* fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// GPU kernel to convert a rgb image to grayscale
__global__ void grayscale_kernel(const std::uint8_t* rgb, std::uint8_t* gray, int width, int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int idx = y * width + x;
    const std::uint8_t& pixel = rgb[idx];
    gray[idx] = 0.21 * pixel.r + 0.71 * pixel.g + 0.07 * pixel.b;
  }
}

void step1(char* hostBuffer, int width, int height, std::ptrdiff_t stride)
{
  hipError_t rc = hipSuccess;

  // Allocate device memory
  char*  devBuffer;
  size_t pitch;

  rc = hipMallocPitch(&devBuffer, &pitch, width * sizeof(rgba8_t), height);
  if (rc)
    abortError("Fail buffer allocation");

  // Run the kernel with blocks of size 64 x 64
  {
    int bsize = 32;
    int w     = std::ceil((float)width / bsize);
    int h     = std::ceil((float)height / bsize);

    spdlog::debug("running kernel of size ({},{})", w, h);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);
    grayscale_kernel<<<dimGrid, dimBlock>>>(devBuffer, width, height, pitch);

    if (hipPeekAtLastError())
      abortError("Computation Error");
  }

  // Copy back to main memory
  rc = hipMemcpy2D(hostBuffer, stride, devBuffer, pitch, width * sizeof(rgba8_t), height, hipMemcpyDeviceToHost);
  if (rc)
    abortError("Unable to copy buffer back to memory");

  // Free
  rc = hipFree(devBuffer);
  if (rc)
    abortError("Unable to free memory");
}
