#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

[[gnu::noinline]] void _abortError(const char *msg, const char *fname, int line)
{
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// GPU kernel to convert a rgb image to grayscale
__global__ void grayscale_kernel(const unsigned char *rgb, unsigned char *gray, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    int idx_offset = idx * channels;
    unsigned char r = rgb[idx_offset];
    unsigned char g = rgb[idx_offset + 1];
    unsigned char b = rgb[idx_offset + 2];
    gray[idx] = static_cast<unsigned char>(0.21 * r + 0.71 * g + 0.07 * b);
}

// GPU kernel to add gaussian blur to an image
// __global__ void gaussian_blur_kernel(const std::uint8_t *src, std::uint8_t *dst, int width, int height)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (x < width && y < height)
//     {
//         int idx = y * width + x;
//         int sum = 0;
//         int count = 0;

//         for (int i = -1; i <= 1; i++)
//         {
//             for (int j = -1; j <= 1; j++)
//             {
//                 int x1 = x + i;
//                 int y1 = y + j;

//                 if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height)
//                 {
//                     sum += src[y1 * width + x1];
//                     count++;
//                 }
//             }
//         }

//         dst[idx] = sum / count;
//     }
// }

// // GPU kernel to compute difference between two images
// __global__ void diff_kernel(const std::uint8_t *img1, const std::uint8_t *img2, std::uint8_t *diff, int width, int height)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (x < width && y < height)
//     {
//         int idx = y * width + x;
//         diff[idx] = abs(img1[idx] - img2[idx]);
//     }
// }

void grayscale_render(unsigned char *rgbBuffer, unsigned char *grayBuffer, int width, int height, int channels)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory
    unsigned char *devBuffer;

    rc = hipMalloc(&devBuffer, width * sizeof(unsigned char) * height);
    if (rc)
        abortError("Fail buffer allocation");

    // Copy image to device
    unsigned char *rgbImage;
    hipMalloc(&rgbImage, width * sizeof(unsigned char) * height * channels);
    rc = hipMemcpy(rgbImage, rgbBuffer, width * sizeof(unsigned char) * height * channels, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail copy image to device");

    // Run the kernel with blocks of size 64 x 64
    {
        int bsize = 32;
        int w = std::ceil((float)width / bsize);
        int h = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);
        // Apply grayscale filter
        grayscale_kernel<<<dimGrid, dimBlock>>>(rgbImage, devBuffer, width, height, channels);

        if (hipPeekAtLastError())
            abortError("Computation Error");
    }

    // Copy back to main memory
    rc = hipMemcpy(grayBuffer, devBuffer, width * sizeof(unsigned char) * height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devBuffer);
    if (rc)
        abortError("Unable to free memory devBuffer");

    rc = hipFree(rgbImage);
    if (rc)
        abortError("Unable to free memory rgbImage");
}
