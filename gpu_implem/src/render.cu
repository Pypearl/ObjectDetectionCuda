#include "hip/hip_runtime.h"
#include "render.hpp"
#include <spdlog/spdlog.h>
#include <cassert>

[[gnu::noinline]] void _abortError(const char *msg, const char *fname, int line)
{
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

// GPU kernel to convert a rgb image to grayscale
__global__ void grayscale_kernel(const unsigned char *rgb, unsigned char *gray, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = y * width + x;
    int idx_offset = idx * channels;
    unsigned char r = rgb[idx_offset];
    unsigned char g = rgb[idx_offset + 1];
    unsigned char b = rgb[idx_offset + 2];
    gray[idx] = static_cast<unsigned char>(0.21 * r + 0.71 * g + 0.07 * b);
}

// GPU kernel to add gaussian blur to an image
__global__ void gaussian_blur_kernel(unsigned char *image, unsigned char *blurImage, int width, int height, float *kernel, int kernelRadius)
{
    const unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (y < height && x < width)
    {
        for (int kx = -kernelRadius; kx <= kernelRadius; kx++)
        {
            for (int ky = -kernelRadius; ky <= kernelRadius; ky++)
            {
                if (y + ky >= 0 && y + ky < height && x + kx >= 0 && x + kx < width)
                {
                    blurImage[y * width + x] += static_cast<unsigned char>(image[(y + ky) * width + (x + kx)] * kernel[kx + kernelRadius] * kernel[ky + kernelRadius]);
                }
            }
        }
    }
}

// // GPU kernel to compute difference between two images
// __global__ void diff_kernel(const std::uint8_t *img1, const std::uint8_t *img2, std::uint8_t *diff, int width, int height)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;

//     if (x < width && y < height)
//     {
//         int idx = y * width + x;
//         diff[idx] = abs(img1[idx] - img2[idx]);
//     }
// }

// Function to render a grayscale image
void grayscale_render(unsigned char *rgbBuffer, unsigned char *grayBuffer, int width, int height, int channels)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory
    unsigned char *devBuffer;

    rc = hipMalloc(&devBuffer, width * sizeof(unsigned char) * height);
    if (rc)
        abortError("Fail buffer allocation");

    // Copy image to device
    unsigned char *devImage;
    hipMalloc(&devImage, width * sizeof(unsigned char) * height * channels);
    rc = hipMemcpy(devImage, rgbBuffer, width * sizeof(unsigned char) * height * channels, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail copy image to device");

    // Run the kernel with blocks of size 64 x 64
    {
        int bsize = 32;
        int w = std::ceil((float)width / bsize);
        int h = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);
        // Apply grayscale filter
        grayscale_kernel<<<dimGrid, dimBlock>>>(devImage, devBuffer, width, height, channels);

        if (hipPeekAtLastError())
            abortError("Computation Error");
    }

    // Copy back to main memory
    rc = hipMemcpy(grayBuffer, devBuffer, width * sizeof(unsigned char) * height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devBuffer);
    if (rc)
        abortError("Unable to free memory devBuffer");

    rc = hipFree(devImage);
    if (rc)
        abortError("Unable to free memory rgbImage");
}

// Function to render a gaussian blur image
void gaussian_blur_render(unsigned char *image, unsigned char *blurImage, int width, int height, float *kernel, int kernelSize)
{
    hipError_t rc = hipSuccess;

    // Allocate device memory
    unsigned char *devBuffer;

    rc = hipMalloc(&devBuffer, width * sizeof(unsigned char) * height);
    if (rc)
        abortError("Fail buffer allocation");

    // Copy image to device
    unsigned char *devImage;
    hipMalloc(&devImage, width * sizeof(unsigned char) * height);
    rc = hipMemcpy(devImage, image, width * sizeof(unsigned char) * height, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail copy image to device");

    // Copy kernel to device
    float *devKernel;
    hipMalloc(&devKernel, kernelSize * sizeof(float) * kernelSize);
    rc = hipMemcpy(devKernel, kernel, kernelSize * sizeof(float) * kernelSize, hipMemcpyHostToDevice);
    if (rc)
        abortError("Fail copy kernel to device");

    // Run the kernel with blocks of size 64 x 64
    {
        int bsize = 32;
        int w = std::ceil((float)width / bsize);
        int h = std::ceil((float)height / bsize);

        spdlog::debug("running kernel of size ({},{})", w, h);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);
        // Apply gaussian blur filter
        gaussian_blur_kernel<<<dimGrid, dimBlock>>>(devImage, devBuffer, width, height, devKernel, kernelSize);

        if (hipPeekAtLastError())
            abortError("Computation Error");
    }

    // Copy back to main memory
    rc = hipMemcpy(blurImage, devBuffer, width * sizeof(unsigned char) * height, hipMemcpyDeviceToHost);
    if (rc)
        abortError("Unable to copy buffer back to memory");

    // Free
    rc = hipFree(devBuffer);
    if (rc)
        abortError("Unable to free memory devBuffer");

    rc = hipFree(devImage);
    if (rc)
        abortError("Unable to free memory devImage");

    rc = hipFree(devKernel);
    if (rc)
        abortError("Unable to free memory devKernel");
}
